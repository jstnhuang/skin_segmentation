#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "Eigen/Dense"
#include "sensor_msgs/Image.h"

#include "skin_segmentation/projection.h"

namespace {
void HandleError(hipError_t err) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__,
           __LINE__);
    exit(EXIT_FAILURE);
  }
}

struct CameraData {
  double inv_depth_fx;
  double inv_depth_fy;
  double depth_cx;
  double depth_cy;
  double depth_Tx;
  double depth_Ty;
};
}

namespace skinseg {
__global__ void gpu_ComputeHandMask(const float4* points, const int height,
                                    const int width, CameraData camera_data,
                                    Eigen::Affine3f* world_in_left,
                                    Eigen::Affine3f* world_in_right,
                                    uint8_t* mask) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row > height - 1 || col > width - 1) {
    return;
  }

  int index = row * width + col;
  float4 point = points[index];
  if (point.w == 0) {
    return;
  }

  Eigen::Vector3f xyz;
  xyz << point.x, point.y, point.z;
  Eigen::Vector3f pos_in_l_frame = *world_in_left * xyz;
  Eigen::Vector3f pos_in_r_frame = *world_in_right * xyz;

  const float min_x = 0.075;
  const float max_x = 0.3;
  const float min_y = -0.12;
  const float max_y = 0.12;
  const float min_z = -0.06;
  const float max_z = 0.06;
  bool in_left_box =
      (pos_in_l_frame.x() > min_x && pos_in_l_frame.x() < max_x &&
       pos_in_l_frame.y() > min_y && pos_in_l_frame.y() < max_y &&
       pos_in_l_frame.z() > min_z && pos_in_l_frame.z() < max_z);
  bool in_right_box =
      (pos_in_r_frame.x() > min_x && pos_in_r_frame.x() < max_x &&
       pos_in_r_frame.y() > min_y && pos_in_r_frame.y() < max_y &&
       pos_in_r_frame.z() > min_z && pos_in_r_frame.z() < max_z);
  mask[index] = in_left_box || in_right_box;
}

void ComputeHandMask(float4* points, int height, int width,
                     const CameraData& camera_data,
                     const Eigen::Affine3f& l_forearm_pose,
                     const Eigen::Affine3f& r_forearm_pose, uint8_t* mask) {
  float4* d_points;
  int points_size = width * height * sizeof(float4);
  HandleError(hipMalloc((void**)&d_points, points_size));
  HandleError(
      hipMemcpy(d_points, points, points_size, hipMemcpyHostToDevice));

  Eigen::Affine3f world_in_left = l_forearm_pose.inverse();
  Eigen::Affine3f* d_l_forearm_pose;
  int pose_size = sizeof(Eigen::Affine3f);
  HandleError(hipMalloc((void**)&d_l_forearm_pose, pose_size));
  HandleError(hipMemcpy(d_l_forearm_pose, world_in_left.data(), pose_size,
                         hipMemcpyHostToDevice));
  Eigen::Affine3f world_in_right = r_forearm_pose.inverse();
  Eigen::Affine3f* d_r_forearm_pose;
  HandleError(hipMalloc((void**)&d_r_forearm_pose, pose_size));
  HandleError(hipMemcpy(d_r_forearm_pose, world_in_right.data(), pose_size,
                         hipMemcpyHostToDevice));

  uint8_t* d_mask;
  int mask_size = height * width * sizeof(uint8_t);
  HandleError(hipMalloc((void**)&d_mask, mask_size));
  HandleError(hipMemset(d_mask, 0, mask_size));

  // Kernel code
  dim3 threadsPerBlock(8, 8);
  dim3 numBlocks(ceil((float)width / threadsPerBlock.x),
                 ceil((float)height / threadsPerBlock.y));
  gpu_ComputeHandMask<<<numBlocks, threadsPerBlock>>>(
      d_points, height, width, camera_data, d_l_forearm_pose, d_r_forearm_pose,
      d_mask);

  HandleError(hipMemcpy(mask, d_mask, mask_size, hipMemcpyDeviceToHost));
  HandleError(hipFree(d_l_forearm_pose));
  HandleError(hipFree(d_r_forearm_pose));
  HandleError(hipFree(d_mask));
  HandleError(hipFree(d_points));
}
}  // namespace skinseg
